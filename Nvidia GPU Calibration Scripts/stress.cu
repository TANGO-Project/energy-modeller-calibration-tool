
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < 30; j++) {
      y[i] = x[i] + y[i];
      y[i] = x[i] - y[i];
    }
    if ( threadIdx.x % 2 == 0 ) {
      y[i] = x[i] + y[i];
    } else {
      y[i] = x[i] - y[i];
    }
  }
}


int main(int argc, char const *argv[])
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory � accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  int blockSize = atoi(argv[2]);
  int numBlocks = atoi(argv[1]);
  printf("The block size is: %d and the number of blocks is: %d and the amount of work is: %d\n\n",blockSize ,numBlocks , N);
  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  //float maxError = 0.0f;
  //for (int i = 0; i < N; i++)
  //  maxError = fmax(maxError, fabs(y[i]-3.0f));
  //std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}